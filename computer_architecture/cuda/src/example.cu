#include "hip/hip_runtime.h"
//本程序用GPGPU计算矩阵乘：A[M,K] * B[K,N] = C[M,N]
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define M 1024
#define N 512
#define K 1024

//kernel函数，这个函数是运行在CUDA核心上的，负责计算 结果矩阵的一个元素
__global__ void multiplyKernel(const int *dev_A, const int *dev_B, int *dev_C) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y; //行
    int Col = blockIdx.x * blockDim.x + threadIdx.x; //列
    int sum = 0;
    for (int k = 0; k < K; k++) sum += dev_A[Row * K + k] * dev_B[k * N + Col];
    dev_C[Row * N + Col] = sum; //C[M,N]
}

//检查C[x,y]计算是否正确
void check_matrix(int x, int y, int *A, int *B, int *C) {
    int sum = 0;
    for (int i = 0; i < K; i++) sum += A[x * K + i] * B[i * N + y];
    if (sum == C[x * N + y])
        printf("C[%d,%d] is right\n", x, y);
    else
        printf("C[%d,%d]=%d, sum=%d\n", x, y, C[x * N + y], sum);
}

// Helper function for using CUDA to multiply matrices in parallel.
hipError_t multiplyWithCuda(int *C, const int *A, const int *B) {
    int *dev_A = 0;
    int *dev_B = 0;
    int *dev_C = 0;
    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Any CUDA-capable GPU installed?");
        goto Error;
    }
    // Allocate GPU buffers(global memory) for three matrices A B C.
    cudaStatus = hipMalloc((void **) &dev_A, M * K * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void **) &dev_B, K * N * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void **) &dev_C, M * N * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    // Copy input matrices from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_A, A, M * K * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_B, B, K * N * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    dim3 dimGrid(16, 32);   //grid的维度
    dim3 dimBlock(32, 32);  //block维度
    // Launch a kernel on the GPU with one thread for each element of C matrix.
    multiplyKernel<<<dimGrid, dimBlock >>>(dev_A, dev_B, dev_C);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d!\n", cudaStatus);
        goto Error;
    }
    // Copy output matrix from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, dev_C, M * N * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    Error:
    hipFree(dev_C);
    hipFree(dev_A);
    hipFree(dev_B);
    return cudaStatus;
}

//初始化矩阵
void init(int rowNum, int colNum, int *matrix) {
    srand((unsigned int) time(0));
    for (int i = 0; i < rowNum * colNum; i++) matrix[i] = rand() % 200;
}

int main() {
    int *A = (int *) malloc(M * K * sizeof(int));  //A[M,K]
    int *B = (int *) malloc(K * N * sizeof(int));  //B[K,N]
    int *C = (int *) malloc(M * N * sizeof(int));  //C[M,N] 结果矩阵
    //initialize matrix A and B
    init(M, K, A);
    init(K, N, B);
    clock_t start_clock, finish_clock; //for timing
    double duration_seconds;
    start_clock = clock();
    hipError_t cudaStatus = multiplyWithCuda(C, A, B);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiplyWithCuda failed!");
        return 1;
    }
    finish_clock = clock();
    duration_seconds = (double) (finish_clock - start_clock) / CLOCKS_PER_SEC;
    printf("CUDA use time (s): %f\n", duration_seconds);
    printf("================================================\n");
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    check_matrix(1023, 511, A, B, C);//检查C[1023,511]是否计算正确
    free(A);
    free(B);
    free(C);
    return 0;
}
