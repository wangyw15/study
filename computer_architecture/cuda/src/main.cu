#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <random>

#include "hip/hip_runtime.h"

#define ARRAY_SIZE 10000

__global__ void ArrayComputeCUDA(int n, const int a[], const int b[], int c[]) {
    for (int i = 0; i < n; i++) {
        // c[i] = a[i] - b[i] * (i % 2 - 1);
        c[i] = a[i] + b[i] * ((i + 1) % 2);
    }
}

void ArrayComputeCPU(int n, const int a[], const int b[], int c[]) {
    for (int i = 0; i < n; i++) {
        if (i % 2 == 0) {
            c[i] = a[i] + b[i];
        }
        else {
            c[i] = a[i];
        }
    }
}

int main() {
    // Get GPU device
    int device = -1;
    auto status = hipGetDevice(&device);
    if (status != hipSuccess) {
        std::cerr << "hipGetDevice failed!" << std::endl;
        return 1;
    }
    std::cout << "Select GPU device: " << device << std::endl;

    // Check GPU
    status = hipSetDevice(device);
    if (status != hipSuccess) {
        std::cerr << "hipSetDevice failed!" << std::endl;
        return 1;
    }

    // Define arrays
    int *a, *b, cpu[ARRAY_SIZE], *gpu;
    hipMallocManaged(&a, ARRAY_SIZE * sizeof(int));
    hipMallocManaged(&b, ARRAY_SIZE * sizeof(int));
    hipMallocManaged(&gpu, ARRAY_SIZE * sizeof(int));

    // Initialize a and b
    std::random_device seed;
    std::mt19937_64 engine(seed());
    std::uniform_int_distribution<int> distribution(0, 100);
    for (int i = 0; i < ARRAY_SIZE; i++) {
        a[i] = distribution(engine);
        b[i] = distribution(engine);
    }

    // Calculate by CPU
    auto start = std::chrono::high_resolution_clock::now();
    ArrayComputeCPU(ARRAY_SIZE, a, b, cpu);
    auto end = std::chrono::high_resolution_clock::now();

    std::cout << "CPU time: "
              << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()
              << "us" << std::endl;

    // Calculate by GPU
    dim3 dimGrid(16, 32);   // grid's dimension
    dim3 dimBlock(32, 32);  // block's dimension

    start = std::chrono::high_resolution_clock::now();
    hipMemPrefetchAsync(a, ARRAY_SIZE * sizeof(int), device);
    hipMemPrefetchAsync(b, ARRAY_SIZE * sizeof(int), device);
    ArrayComputeCUDA<<<dimGrid, dimBlock>>>(ARRAY_SIZE, a, b, gpu);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();

    std::cout << "GPU time: "
              << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()
              << "us" << std::endl;

    // Check the result
    for (int i = 0; i < ARRAY_SIZE; i++) {
        if (cpu[i] != gpu[i]) {
            std::cerr << "Error at " << i << ": " << cpu[i] << " != " << gpu[i] << std::endl;
            return 1;
        }
    }
    std::cout << "All results are correct!" << std::endl;

    return 0;
}
