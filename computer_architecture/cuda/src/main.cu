#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <random>

#include "hip/hip_runtime.h"

#define ARRAY_SIZE 10000

__global__ void ArrayComputeCUDA(int n, const int a[], const int b[], int c[]) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] - b[i] * (i % 2 - 1);
    }
}

void ArrayComputeCPU(int n, const int a[], const int b[], int c[]) {
    for (int i = 0; i < n; i++) {
        // c[i] = a[i] - b[i] * (i % 2 - 1);
        if (i % 2 == 0) {
            c[i] = a[i] + b[i];
        }
        else {
            c[i] = a[i];
        }
    }
}

int main() {
    int device = 1;
    hipGetDevice(&device);
    // Check GPU
    auto status = hipSetDevice(device);
    if (status != hipSuccess) {
        std::cerr << "hipSetDevice failed! Any CUDA-capable GPU installed?" << std::endl;
        return 1;
    }

    // Define arrays
    int *a, *b, cpu[ARRAY_SIZE], *gpu;
    hipMallocManaged(&a, ARRAY_SIZE * sizeof(int));
    hipMallocManaged(&b, ARRAY_SIZE * sizeof(int));
    hipMallocManaged(&gpu, ARRAY_SIZE * sizeof(int));

    // Initialize a and b
    std::random_device seed;
    std::mt19937_64 engine(seed());
    std::uniform_int_distribution<int> distribution(0, 100);
    for (int i = 0; i < ARRAY_SIZE; i++) {
        a[i] = distribution(engine);
        b[i] = distribution(engine);
    }

    // Calculate by CPU
    auto start = std::chrono::high_resolution_clock::now();
    ArrayComputeCPU(ARRAY_SIZE, a, b, cpu);
    auto end = std::chrono::high_resolution_clock::now();

    std::cout << "CPU time: "
              << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()
              << "ms" << std::endl;

    // Calculate by GPU
    dim3 dimGrid(16, 32);   // grid's dimension
    dim3 dimBlock(32, 32);  // block's dimension

    start = std::chrono::high_resolution_clock::now();
    hipMemPrefetchAsync(a, ARRAY_SIZE * sizeof(int), device);
    hipMemPrefetchAsync(b, ARRAY_SIZE * sizeof(int), device);
    ArrayComputeCUDA<<<dimGrid, dimBlock>>>(ARRAY_SIZE, a, b, gpu);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();

    std::cout << "GPU time: "
              << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()
              << "ms" << std::endl;

    // Check the result
    for (int i = 0; i < ARRAY_SIZE; i++) {
        if (cpu[i] != gpu[i]) {
            std::cerr << "Error at " << i << ": " << cpu[i] << " != " << gpu[i] << std::endl;
            return 1;
        }
    }
    std::cout << "All results are correct!" << std::endl;

    return 0;
}
